#include "hip/hip_runtime.h"
#include "functions_cuda.h"

void transferCosts(Matrix *d_y_costs_dev, Matrix *d_x_costs_dev, Vertices *d_vertices_dev, int N, int K, unsigned int devid, int *DSPC_x, int *DSPC_y, int offset_y, int offset_x)
{

	cudaSafeCall(hipSetDevice(devid), "Error in hipSetDevice function_cuda::initializeYCosts");

	dim3 blocks_per_grid;
	dim3 threads_per_block;
	int total_blocks = 0;
	int y_size = N * N * N;
	calculateRectangularDims(blocks_per_grid, threads_per_block, total_blocks, y_size, DSPC_y[devid]);
	kernel_transferCosts_cuda<<<blocks_per_grid, threads_per_block>>>(d_y_costs_dev[devid].elements, d_x_costs_dev[devid].elements, d_vertices_dev[devid].row_duals, d_vertices_dev[devid].col_duals, devid, N, K, DSPC_y[devid], DSPC_x[devid], offset_y, offset_x);
	hipDeviceSynchronize(); // was required to make the code enter the kernel
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
	}
	cudaSafeCall_new(hipGetLastError(), "Error in kernel_initializeYCosts Functions initializeYCosts");
}

void multiplier_update(Matrix *d_y_costs_dev, int N, int K, unsigned int devid, int *DSPC_y, int offset_y, int devcount, int procid, int procsize)
{

	cudaSafeCall(hipSetDevice(devid), "Error in hipSetDevice function_cuda::initializeYCosts");
	dim3 blocks_per_grid;
	dim3 threads_per_block;
	int total_blocks = 0;
	int y_size = N * N;

	calculateRectangularDims(blocks_per_grid, threads_per_block, total_blocks, y_size, DSPC_y[devid]);
	kernel_multiplier_update_cuda<<<blocks_per_grid, threads_per_block>>>(d_y_costs_dev[devid].elements, N, K, devid, DSPC_y[devid], offset_y, devcount, procid, procsize);
	hipDeviceSynchronize(); // was required to make the code enter the kernel
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
	}
	cudaSafeCall_new(hipGetLastError(), "Error in kernel_initializeYCosts Functions initializeYCosts");
}

void solveYLSAP(Matrix *d_y_costs_dev, Matrix *d_x_costs_dev, int N, int K, unsigned int devid, int *DSPC_x, int *DSPC_y, int offset_y, int offset_x)
{

	cudaSafeCall(hipSetDevice(devid), "Error in hipSetDevice function_cuda::initializeYCosts");
	dim3 blocks_per_grid;
	dim3 threads_per_block;
	int total_blocks = 0;
	// int y_size = N * N * N;
	// int x_size = N * N;
	// printf("%d\n", DSPC_y[devid]);

	calculateRectangularDims(blocks_per_grid, threads_per_block, total_blocks, N, N);
	for (std::size_t i = 0; i < DSPC_y[devid]; i++)
	{
		kernel_solveYLSAP_cuda_min<<<blocks_per_grid, threads_per_block>>>(d_y_costs_dev[devid].elements, d_x_costs_dev[devid].elements, devid, N, K, DSPC_y[devid], DSPC_x[devid], offset_y, offset_x, i);

		cudaSafeCall_new(hipGetLastError(), "Error in kernel_initializeYCosts Functions initializeYCosts");
	}

	for (int i = 0; i < DSPC_y[devid]; i++)
	{
		kernel_solveYLSAP_cuda_dual<<<blocks_per_grid, threads_per_block>>>(d_y_costs_dev[devid].elements, d_x_costs_dev[devid].elements, devid, N, K, DSPC_y[devid], DSPC_x[devid], offset_y, offset_x, i);
		cudaSafeCall_new(hipGetLastError(), "Error in kernel_initializeYCosts Functions initializeYCosts");
	}

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
	}
	cudaSafeCall_new(hipGetLastError(), "Error in kernel_initializeYCosts Functions initializeYCosts");
}

// void solveYLSAP(Matrix *d_y_costs_dev, Matrix *d_x_costs_dev, int N , int K, unsigned int devid, int *DSPC_x, int *DSPC_y, int offset_y, int offset_x){
//
// 		cudaSafeCall(hipSetDevice(devid), "Error in hipSetDevice function_cuda::initializeYCosts");
// 		dim3 blocks_per_grid;
// 		dim3 threads_per_block;
// 		int total_blocks = 0;
// 		int y_size = N * N * N;
// 		int x_size = N * N;
//
// 		calculateRectangularDims(blocks_per_grid, threads_per_block, total_blocks, x_size, DSPC_y[devid]);
//
// 		hipDeviceSynchronize(); //was required to make the code enter the kernel
// 		hipError_t error = hipGetLastError();
// 		if (error != hipSuccess) {
// 		fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
// 		}
// 		cudaSafeCall(hipGetLastError(), "Error in kernel_initializeYCosts Functions initializeYCosts");
// }

void computeUB(Matrix *d_x_costs_dev, Matrix *d_y_costs_dev, Vertices *d_vertices_dev, int N, int K, Objective *d_UB_dev, unsigned int devid, int *DSPC_x)
{

	cudaSafeCall(hipSetDevice(devid), "Error in hipSetDevice function_cuda::initializeYCosts");

	dim3 blocks_per_grid;
	dim3 threads_per_block;
	int total_blocks = 0;
	int x_size = N;
	calculateRectangularDims(blocks_per_grid, threads_per_block, total_blocks, x_size, DSPC_x[devid]);
	kernel_computeUB<<<blocks_per_grid, threads_per_block>>>(d_y_costs_dev[devid].elements, d_x_costs_dev[devid].elements, d_vertices_dev[devid].row_assignments, N, K, DSPC_x[devid], d_UB_dev[devid].obj);
	// hipDeviceSynchronize(); //was required to make the code enter the kernel
	//  printDebugArray(d_y_costs_dev[devid].elements, DSPC_y[devid] * N * N * N, 	"0", devid);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
	}
	cudaSafeCall(hipGetLastError(), "Error in kernel_initializeYCosts Functions initializeYCosts");
}

__global__ void kernel_computeUB(double *d_y_costs, double *d_x_costs, int *d_row_assignments, int N, int K, int SP, double *d_UB)
{

	int p = blockIdx.y * blockDim.y + threadIdx.y;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (p < SP - 1 && i < N)
	{
		int j = d_row_assignments[N * p + i] % N;
		int k = d_row_assignments[N * (p + 1) + j] % N;

		atomicAdd(&d_UB[p], d_x_costs[p * N * N + N * i + j] + d_y_costs[N * N * N * p + N * N * i + N * j + k]);
		// d_UB[p] += d_x_costs[ p * N * N + N * i + j];
		// 	d_UB[p] += d_y_costs[N * N * N * p + N * N * i + N * j +k] ;
		// printf("%f\n",  d_x_costs[ p * N * N + N * i + j]);
	}
}

__global__ void kernel_transferCosts_cuda(double *d_y_costs, double *d_x_costs, double *d_row_duals, double *d_col_duals, unsigned int devid, std::size_t N, std::size_t K, int DSPC_y, int DSPC_x, int offset_y, int offset_x)
{

	std::size_t ylapid = blockIdx.y * blockDim.y + threadIdx.y;
	std::size_t ijk = blockIdx.x * blockDim.x + threadIdx.x;
	if (ylapid < DSPC_y)
	{

		std::size_t i = ijk / (N * N);
		std::size_t j = ijk % (N * N) / N;
		std::size_t k = ijk % N;

		if (i < N && j < N && k < N)
		{
			d_y_costs[(ylapid * N * N * N) + N * N * k + N * i + j] += d_x_costs[(ylapid) * (N * N) + N * i + j] - d_row_duals[(ylapid) * (N) + i] - d_col_duals[(ylapid) * (N) + j];
		}
	}
}

__global__ void kernel_multiplier_update_cuda(double *d_y_costs, std::size_t N, std::size_t K, unsigned int devid, int DSPC_y, int offset_y, int devcount, int procid, int procsize)
{

	std::size_t ylapid = blockIdx.y * blockDim.y + threadIdx.y;
	std::size_t ijk = blockIdx.x * blockDim.x + threadIdx.x;
	ylapid = ylapid * 2;
	if (devid != devcount - 1)
	{
		//	if(DSPC_y %2 ==0){

		if (ylapid < DSPC_y)
		{

			std::size_t j = ijk / N;
			std::size_t k = ijk % N;
			std::size_t y_size = N * N * N;

			if (ylapid < K - 2 && j < N && k < N)
			{

				double sum = 0;

				double min_cost1 = INF;
				double min_cost2 = INF;
				std::size_t k1 = 0;
				std::size_t k2 = 0;

				/////////////////////////////////////////////////////////////////////////

				for (std::size_t i_ = 0; i_ < N; i_++)
				{
					std::size_t y_id1 = (ylapid + 1) * y_size + N * N * i_ + N * j + k;
					if (min_cost1 >= d_y_costs[y_id1])
					{
						min_cost1 = d_y_costs[y_id1];
						k1 = i_;
					}
				}

				/////////////////////////////////////////////////////////////////////////

				for (std::size_t k_ = 0; k_ < N; k_++)
				{
					std::size_t y_id2 = (ylapid)*y_size + N * N * k + N * k_ + j;
					if (min_cost2 >= d_y_costs[y_id2])
					{
						min_cost2 = d_y_costs[y_id2];
						k2 = k_;
					}
				}
				/////////////////////////////////////////////////////////////////////////

				sum = (min_cost1) + (min_cost2);

				std::size_t yid1 = (ylapid + 1) * y_size + N * N * k1 + N * j + k;
				std::size_t yid2 = (ylapid)*y_size + N * N * k + N * k2 + j;

				d_y_costs[yid1] = sum * 0.44;

				d_y_costs[yid2] = sum * 0.56;

				//  d_y_costs[yid1] = sum * 0.5;
				// d_y_costs[yid2] = sum * 0.5;
			}
		}
	}

	if (devid == devcount - 1)
	{
		//	if(DSPC_y%2!=0){
		if (ylapid < DSPC_y - 1)
		{

			std::size_t j = ijk / N;
			std::size_t k = ijk % N;
			std::size_t y_size = N * N * N;

			if (ylapid < K - 2 && j < N && k < N)
			{

				double sum = 0;

				double min_cost1 = INF;
				double min_cost2 = INF;
				std::size_t k1 = 0;
				std::size_t k2 = 0;

				/////////////////////////////////////////////////////////////////////////

				for (std::size_t i_ = 0; i_ < N; i_++)
				{
					std::size_t y_id1 = (ylapid + 1) * y_size + N * N * i_ + N * j + k;
					if (min_cost1 >= d_y_costs[y_id1])
					{
						min_cost1 = d_y_costs[y_id1];
						k1 = i_;
					}
				}

				/////////////////////////////////////////////////////////////////////////

				for (std::size_t k_ = 0; k_ < N; k_++)
				{
					std::size_t y_id2 = (ylapid)*y_size + N * N * k + N * k_ + j;
					if (min_cost2 >= d_y_costs[y_id2])
					{
						min_cost2 = d_y_costs[y_id2];
						k2 = k_;
					}
				}
				/////////////////////////////////////////////////////////////////////////

				sum = (min_cost1) + (min_cost2);

				std::size_t yid1 = (ylapid + 1) * y_size + N * N * k1 + N * j + k;
				std::size_t yid2 = (ylapid)*y_size + N * N * k + N * k2 + j;

				d_y_costs[yid1] = sum * 0.44;

				d_y_costs[yid2] = sum * 0.56;

				//  d_y_costs[yid1] = sum * 0.5;
				// d_y_costs[yid2] = sum * 0.5;
			}
		}
	}
}

__global__ void kernel_solveYLSAP_cuda_min(double *d_y_costs, double *d_x_costs, unsigned int devid, std::size_t N, std::size_t K, int DSPC_y, int DSPC_x, int offset_y, int offset_x, std::size_t ylapid)
{

	// int ylapid = blockIdx.y * blockDim.y + threadIdx.y;

	std::size_t i = blockIdx.y * blockDim.y + threadIdx.y;
	std::size_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N && j < N)
	{

		double min = INF;
		for (std::size_t k = 0; k < N; k++)
		{
			double cost = d_y_costs[ylapid * N * N * N + k * N * N + i * N + j];
			if (cost < min)
				min = cost;
		}

		d_x_costs[ylapid * N * N + i * N + j] = min;
	}
}
__global__ void kernel_solveYLSAP_cuda_dual(double *d_y_costs, double *d_x_costs, unsigned int devid, std::size_t N, std::size_t K, int DSPC_y, int DSPC_x, int offset_y, int offset_x, std::size_t ylapid)
{

	// int ylapid = blockIdx.y * blockDim.y + threadIdx.y;

	std::size_t i = blockIdx.y * blockDim.y + threadIdx.y;
	std::size_t j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N && j < N)
	{

		double min = d_x_costs[ylapid * N * N + i * N + j];
		for (std::size_t k = 0; k < N; k++)
		{
			d_y_costs[ylapid * N * N * N + k * N * N + i * N + j] -= min;
		}
	}
}
